#include "hip/hip_runtime.h"
/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <hipcub/hipcub.hpp>

__global__ void sortOutput(
    int* d_indexes, float* d_scores, float* d_boxes, int* d_classes, float* bboxData, float* scoreData,
    const uint numOutputClasses)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    int index = d_indexes[x_id];
    int maxIndex = d_classes[index];
    bboxData[x_id * 4 + 0] = d_boxes[index * 4 + 0];
    bboxData[x_id * 4 + 1] = d_boxes[index * 4 + 1];
    bboxData[x_id * 4 + 2] = d_boxes[index * 4 + 2];
    bboxData[x_id * 4 + 3] = d_boxes[index * 4 + 3];
    scoreData[x_id * numOutputClasses + maxIndex] = d_scores[x_id] - 1.f;
}

hipError_t sortDetections(
    void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* bboxData, void* scoreData, void* countData,
    const uint& batchSize, uint64_t& outputSize, uint& topK, const uint& numOutputClasses, hipStream_t stream);

hipError_t sortDetections(
    void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* bboxData, void* scoreData, void* countData,
    const uint& batchSize, uint64_t& outputSize, uint& topK, const uint& numOutputClasses, hipStream_t stream)
{
    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        int* _d_indexes = reinterpret_cast<int*>(d_indexes) + (batch * outputSize);
        float* _d_scores = reinterpret_cast<float*>(d_scores) + (batch * outputSize);

        int* _countData = reinterpret_cast<int*>(countData) + (batch);
        int* _count = (int*)malloc(sizeof(int));
        hipMemcpy(_count, (int*)&_countData[0], sizeof(int), hipMemcpyDeviceToHost);
        int count = _count[0];

        if (count == 0)
        {
            free(_count);
            return hipGetLastError();
        }

        size_t begin_bit = 0;
        size_t end_bit = sizeof(float) * 8;

        float *d_keys_out = NULL;
        int *d_values_out = NULL;

        hipMalloc((void **)&d_keys_out, count * sizeof(float));
        hipMalloc((void **)&d_values_out, count * sizeof(int));

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;

        hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, _d_scores, d_keys_out, _d_indexes,
        d_values_out, count, begin_bit, end_bit);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, _d_scores, d_keys_out, _d_indexes,
        d_values_out, count, begin_bit, end_bit);

        hipMemcpy(_d_scores, d_keys_out, count * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(_d_indexes, d_values_out, count * sizeof(int), hipMemcpyDeviceToDevice);

        int threads_per_block = count < topK ? count : topK;

        sortOutput<<<1, threads_per_block, 0, stream>>>(
            _d_indexes, _d_scores, reinterpret_cast<float*>(d_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_classes) + (batch * outputSize), reinterpret_cast<float*>(bboxData) + (batch * topK),
            reinterpret_cast<float*>(scoreData) + (batch * topK), numOutputClasses);

        hipFree(d_keys_out);
        hipFree(d_values_out);
        hipFree(d_temp_storage);

        free(_count);
    }
    return hipGetLastError();
}
